#include "hip/hip_runtime.h"
#include <cassert>
#include <cfloat>
#include "private.h"

__constant__ uint32_t d_dim;

__global__ void gamma_cuda(uint32_t size, const float *__restrict__ v1, float *v2) {
  uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= size) {
    return;
  }
  v2[index] = -logf(v1[index] * v2[index]);
}

__global__ void log_cuda(uint32_t size, float *v) {
  uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= size) {
    return;
  }
  v[index] = logf(v[index]);
}

/*
  weights, cols, rows - CSR format
  size - number of matrix rows; rows array contains (size + 1) elements
  plan - execution plan, consists of 2 parts: first is offset table and
         the second is the row indices
*/
__global__ void weighted_minhash_cuda(
    const float *__restrict__ rs, const float *__restrict__ ln_cs,
    const float *__restrict__ betas, const float *__restrict__ weights,
    const uint32_t *__restrict__ cols, const uint32_t *__restrict__ rows,
    const int32_t *__restrict__ plan, const int sample_delta,
    uint32_t *__restrict__ hashes) {
  const uint32_t thread_index = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t sample_index = threadIdx.y;
  int32_t row_offset = plan[thread_index];
  int32_t row_border = plan[thread_index + 1];
  if (row_offset == row_border) {
    return;
  }
  const uint32_t sample_offset = sample_index * sample_delta;
  const uint32_t samples = blockDim.y * sample_delta;
  extern __shared__ float shmem[];
  float *lnmins = &shmem[(threadIdx.x * blockDim.y + sample_index) * 3 * sample_delta];
  uint32_t *dmins = reinterpret_cast<uint32_t* >(lnmins + sample_delta);
  uint32_t *tmins = dmins + sample_delta;
  int32_t row = -1;
  uint32_t border = 0, index = 0;
  for (;; index++) {
    if (index >= border) {
      for (uint32_t s = 0; s < sample_delta; s++) {
        lnmins[s] = FLT_MAX;
      }
      if (row >= 0) {
        for (int s = 0; s < sample_delta; s++) {
          auto hash = hashes + (row * samples + s + sample_offset) * 2;
          hash[0] = dmins[s];
          hash[1] = tmins[s];
        }
      }
      if (row_offset >= row_border) {
        break;
      }
      row = plan[row_offset++];
      index = rows[row];
      border = rows[row + 1];
    }
    float w = logf(weights[index]);
    float d = cols[index];
    #pragma unroll 4
    for (int s = 0; s < sample_delta; s++) {
      int64_t ci = s + sample_offset; ci *= d_dim; ci += d;
      float r = rs[ci];
      float beta = betas[ci];
      float t = floorf(w / r + beta);
      float ln_y = (t - beta) * r;
      float ln_a = ln_cs[ci] - ln_y - r;
      if (ln_a < lnmins[s]) {
        lnmins[s] = ln_a;
        dmins[s] = d;
        tmins[s] = t;
      }
    }
  }
}

extern "C" {

hipError_t gamma_(uint32_t size, const float *v1, float *v2) {
  dim3 block(1024, 1, 1);
  dim3 grid(size / block.x + 1, 1, 1);
  gamma_cuda<<<grid, block>>>(size, v1, v2);
  RETERR(hipDeviceSynchronize());
  return hipSuccess;
}

hipError_t log_(uint32_t size, float *v) {
  dim3 block(1024, 1, 1);
  dim3 grid(size / block.x + 1, 1, 1);
  log_cuda<<<grid, block>>>(size, v);
  RETERR(hipDeviceSynchronize());
  return hipSuccess;
}

MHCUDAResult setup_weighted_minhash(uint32_t dim, int verbosity) {
  CUCH(hipMemcpyToSymbol(HIP_SYMBOL(d_dim), &dim, sizeof(dim)),
       mhcudaMemoryCopyError);
  return mhcudaSuccess;
}

MHCUDAResult weighted_minhash(
    const udevptrs<float> &rs, const udevptrs<float> &ln_cs,
    const udevptrs<float> &betas, const udevptrs<float> &weights,
    const udevptrs<uint32_t> &cols, const udevptrs<uint32_t> &rows,
    int samples, const std::vector<int> &sample_deltas,
    const udevptrs<int32_t> &plan, const std::vector<uint32_t> &grid_sizes,
    const std::vector<int> &devs, int verbosity, udevptrs<uint32_t> *hashes) {
  FOR_EACH_DEVI(
    int sample_delta = sample_deltas[devi];
    int spt = samples / sample_delta;
    assert(MINHASH_BLOCK_SIZE % spt == 0);
    dim3 block(MINHASH_BLOCK_SIZE / spt, spt, 1);
    dim3 grid(grid_sizes[devi], 1, 1);
    auto shmem = 3 * 4 * MINHASH_BLOCK_SIZE * sample_delta;
    DEBUG("dev #%d: <<<%d, [%d, %d], %d>>>\n", devs[devi], grid.x, block.x, block.y, shmem);
    weighted_minhash_cuda<<<grid, block, shmem>>>(
        rs[devi].get(), ln_cs[devi].get(), betas[devi].get(),
        weights[devi].get(), cols[devi].get(), rows[devi].get(),
        plan[devi].get(), sample_delta, (*hashes)[devi].get());
  );
  return mhcudaSuccess;
}

}  // extern "C"